#include "hip/hip_runtime.h"
/* Udacity HW5
  Histogramming for Speed

  The goal of this assignment is compute a histogram
  as fast as possible.  We have simplified the problem as much as
  possible to allow you to focus solely on the histogramming algorithm.

  The input values that you need to histogram are already the exact
  bins that need to be updated.  This is unlike in HW3 where you needed
  to compute the range of the data and then do:
  bin = (val - valMin) / valRange to determine the bin.

  Here the bin is just:
  bin = val

  so the serial histogram calculation looks like:
  for (i = 0; i < numElems; ++i)
   histo[val[i]]++;

  That's it!  Your job is to make it run as fast as possible!

  1. Пользуемся распределением
  // Центральные значения обновляются много чаще
  The values are normally distributed - you may take
  advantage of this fact in your implementation.

  2. Базовая стратегия. 
  http://forums.udacity.com/questions/100040006/any-hw5-timing-results-on-a-solution-that-doesnt-simply-use-atomicadd-in-shared-and-global-mem#cs344
  Не ясно что с сортировкой - это же тоже время. и что сортировать то?
  И если много монипуляций, то может и не выйдет быстрее

*/


#include "utils.h"


#include <cstdio>
static const int g_chunk = 8;
//__device__ __constant__   // падает произв
static const int d_chunk = g_chunk;

//static 
__global__ void src_histo_kernel(
    const unsigned int * const d_vals,
          unsigned int * const d_histo, 
    const unsigned int numBins,
    const unsigned int numElems)
{ 
  int g_id = threadIdx.x + blockDim.x * blockIdx.x;
  if (g_id >= numElems)
    return; 

  // bin
  unsigned int bin = d_vals[g_id];

  // Inc global memory. Partial histos not used.
  atomicAdd(&(d_histo[bin]), 1);
}

__global__
void yourHisto(const unsigned int* const vals, //INPUT
               unsigned int* const histo,      //OUPUT
               int numVals)
{
  //TODO fill in this kernel to calculate the histogram
  //as quickly as possible

  //Although we provide only one kernel skeleton,
  //feel free to use more if it will help you
  //write faster code
}

// Размер блока должен быть не меньше числа бинов
// 
static __global__ 
void histo_kernel_bins_shared_any_any(
    const unsigned int * const d_vals,
          unsigned int * const d_histo, 
    const unsigned int numBins,
    const unsigned int numElems)    
{ 
  extern   // если захардкодить размер, то быстрее не будет
  __shared__ unsigned int local_histo[];
  //__shared__ 
  

  int g_id = blockDim.x * blockIdx.x + threadIdx.x;
  if (g_id >= numElems)
    return; 

  int ref = g_id * d_chunk;
  //if (ref+d_chunk >= numElems);
  //  return;


  for (int i = threadIdx.x; i < numBins; i += blockDim.x) 
    local_histo[i] = 0;

  unsigned slice[d_chunk];
  for (int i = 0; i < d_chunk; ++i) {
    // bin
    const unsigned bin = d_vals[ref+i];  // read is coal.
    slice[i] = bin;
  }

  __syncthreads();

  // каждый поток обрабатывает отрезок
  for (int i = 0; i < d_chunk; ++i) {
    // bin
    //const unsigned bin = d_vals[ref+i];  // read is coal.
    const int bin = slice[i];

    // Inc global memory. Partial histos not used.
    // если поток столько же скольно и бинов, то гонок не будет
    //++local_histo[bin];  // значения могут быть одинаковыми
    atomicAdd(&(local_histo[bin]), 1);
  }

  __syncthreads();  // ждем пока посчитают все потоки

  // Только первые потоки блока обновляеют гистограмму
  for (int i = threadIdx.x; i < numBins; i += blockDim.x) {
    const unsigned int value = local_histo[i];
      if (value)  // уменьшило время но не на много
        // 1:10000
        atomicAdd(&(d_histo[i]), value);  // write is coal., но большая конкуренция
  }
}

static __global__ 
void histo_kernel_bins_shared_any_any_no_reduce(
    const unsigned int * const d_vals,
          unsigned int * const d_histos, 
    const unsigned int numBins,
    const unsigned int numElems)    
{ 
  extern   // если захардкодить размер, то быстрее не будет
  __shared__ unsigned int local_histo[];
  //__shared__ 
  

  int g_id = blockDim.x * blockIdx.x + threadIdx.x;
  if (g_id >= numElems)
    return; 

  int ref = g_id * d_chunk;
  //if (ref+d_chunk >= numElems);
  //  return;


  for (int i = threadIdx.x; i < numBins; i += blockDim.x) 
    local_histo[i] = 0;

  unsigned slice[d_chunk];
  for (int i = 0; i < d_chunk; ++i) {
    // bin
    const unsigned bin = d_vals[ref+i];  // read is coal.
    slice[i] = bin;
  }
  __syncthreads();

  // каждый поток обрабатывает отрезок
  for (int i = 0; i < d_chunk; ++i) {
    const int bin = slice[i];
    atomicAdd(&(local_histo[bin]), 1);
  }  
  __syncthreads(); 

  for (int i = threadIdx.x; i < numBins; i += blockDim.x) {
    d_histos[i + numBins * blockIdx.x] = local_histo[i];
  }
}

static __global__ 
void histo_kernel_bins_shared_any_tblock(
    const unsigned int * const d_vals,
          unsigned int * const d_histo, 
    const unsigned int numBins,
    const unsigned int numElems)    
{ 
  extern   // если захардкодить размер, то быстрее не будет
  __shared__ unsigned int local_histo[];

  int g_id = blockDim.x * blockIdx.x + threadIdx.x;
  //int tid = threadIdx.x;
  if (g_id >= numElems)
    return; 


  for (int i = threadIdx.x; i < numBins; i += blockDim.x) 
    local_histo[i] = 0;

  __syncthreads();

  // каждый поток обрабатывает отрезок
  //if (threadIdx.x % 8 == 0)
    //for (int i = 0, end = 1; i < end; ++i) {
      // bin
      unsigned int bin = d_vals[g_id+0];  // read is coal.

      // Inc global memory. Partial histos not used.
      // если поток столько же скольно и бинов, то гонок не будет
      //++local_histo[bin];  // значения могут быть одинаковыми
      atomicAdd(&(local_histo[bin]), 1);
    //}

  __syncthreads();  // ждем пока посчитают все потоки

  // Только первые потоки блока обновляеют гистограмму
  for (int i = threadIdx.x; i < numBins; i += blockDim.x) {
    const unsigned int value = local_histo[i];
      if (value)  // уменьшило время но не на много
        atomicAdd(&(d_histo[i]), value);  // write is coal.
  }
}

static __global__ 
void resude_kernel(
    const unsigned int * const d_vals,
          unsigned int * const d_histo, 
    const unsigned int numBins,
    const unsigned int numElems) 
  {
  
  for (int j = 0; j < numBins; ++j) {
    int sum = 0;
    for (int i = j; ; i += numBins) {
      if (i > numElems)
        break;
      sum += d_vals[i];  
    }
    d_histo[j] = sum;
  }
}

void computeHistogram(const unsigned int* const d_vals, //INPUT
                            unsigned int* const d_histo,      //OUTPUT
                      const unsigned int numBins,
                      const unsigned int numElems)
{
  //TODO Launch the yourHisto kernel
  // перемешанность данных балансирует нагрузку - если отсортировать вход - производительность резко упадет
  assert(numBins == 1024);
  //const int maxThreadsPerBlock = 1024;

	int threads = 1024;  // пробую меньше, чем число бинов
  int blocks = ceil((1.0f*numElems) / threads);
  printf("blocks: %d\n", blocks);
  //blocks /= 8;
  //int sub_hist_blocks = ceil((1.0f*numElems) / numBins);

  // и блок то меньше не сделать - будет не хватать индексов
  //assert(threads >= numBins);

  //thrust::sort(d_vals, d_vals + numElems);
  // Если отсортировать, то бины которые по середине будут писать практически в одно место - это ограничивает конкуренцию

  unsigned int *d_vals_;
  checkCudaErrors(hipMalloc(&d_vals_,    sizeof(unsigned int) * numElems));
  //if you want to use/launch more than one kernel,
  //feel free
  //src_histo_kernel<<< blocks, threads >>>(d_vals, d_histo, numBins, numElems);
  // 
  // Память может быть большей чем размеры блоков, просто правильно нужно будет обновить глоб. гист.
  //histo_kernel_bins_shared_any_tblock
  //histo_kernel_bins_shared_any_any
  histo_kernel_bins_shared_any_any_no_reduce
  <<< 
    //blocks, threads/g_chunk, (numBins)* sizeof(unsigned int) /*+ 225*/ >>>
    blocks, threads/g_chunk, (numBins+4)* sizeof(unsigned int) + 225 >>>
    (d_vals, 
    //d_histo, 
      d_vals_,
    numBins, numElems);
  //histo_kernel_bins_shared<<< sub_hist_blocks, numBins, numBins * sizeof(unsigned int) >>>(d_vals, d_histo, numBins, numElems);
  resude_kernel<<<1, 1>>>(d_vals_, d_histo, numBins, numElems);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  hipFree(d_vals_);
}
